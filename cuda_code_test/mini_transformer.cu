
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>


const int M = 4;  // rows in X
const int D = 4;  // dim

__global__ void matmul(float* A, float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < M && col < N) {
        float sum = 0;
        for (int i = 0; i < K; i++)
            sum += A[row * K + i] * B[i * N + col];
        C[row * N + col] = sum;
    }
}

float* transpose(float* mat, int rows, int cols) {
    float* trans = new float[rows * cols];
    for (int r = 0; r < rows; r++)
        for (int c = 0; c < cols; c++)
            trans[c * rows + r] = mat[r * cols + c];
    return trans;
}

void gpu_matmul(float* A, float* B, float* C, int M, int N, int K) {
    float *d_A, *d_B, *d_C;
    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);

    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + 15) / 16, (M + 15) / 16);
    matmul<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, M, N, K);

    hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
}

void print_matrix(const char* name, float* mat, int rows, int cols) {
    std::cout << name << ":\n";
    for (int r = 0; r < rows; ++r) {
        for (int c = 0; c < cols; ++c)
            std::cout << mat[r * cols + c] << "\t";
        std::cout << "\n";
    }
    std::cout << "\n";
}

int main() {
    srand(time(0));

    // Allocate host matrices
    float *X = new float[M * D];
    float *Wq = new float[D * D], *Wk = new float[D * D], *Wv = new float[D * D];
    float *Q = new float[M * D], *K = new float[M * D], *V = new float[M * D];
    float *A = new float[M * M], *O = new float[M * D];

    // Randomly initialize
    for (int i = 0; i < M * D; ++i) {
        X[i] = static_cast<float>(rand() % 10);
    }

    for (int i = 0; i < D * D; ++i) {
        Wq[i] = static_cast<float>(rand() % 10);
        Wk[i] = static_cast<float>(rand() % 10);
        Wv[i] = static_cast<float>(rand() % 10);
    }

    // Multiply: Q = X × Wq, K = X × Wk, V = X × Wv
    gpu_matmul(X, Wq, Q, M, D, D);
    gpu_matmul(X, Wk, K, M, D, D);
    gpu_matmul(X, Wv, V, M, D, D);

    // Transpose K → K_T
    float* K_T = transpose(K, M, D);

    // Multiply: A = Q × Kᵗ (M×D × D×M = M×M)
    gpu_matmul(Q, K_T, A, M, M, D);

    // Multiply: O = A × V (M×M × M×D = M×D)
    gpu_matmul(A, V, O, M, D, M);

    // Display results
    print_matrix("X", X, M, D);
    print_matrix("Wq", Wq, D, D);
    print_matrix("Q = X × Wq", Q, M, D);
    print_matrix("A = Q × Kᵗ", A, M, M);
    print_matrix("Output = A × V", O, M, D);

    // Cleanup
    delete[] X; delete[] Wq; delete[] Wk; delete[] Wv;
    delete[] Q; delete[] K; delete[] V;
    delete[] A; delete[] O; delete[] K_T;

    return 0;
}
